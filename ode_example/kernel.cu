#include "hip/hip_runtime.h"
#pragma warning(disable:4819)

#ifndef _CRT_SECURE_NO_WARNINGS
#define _CRT_SECURE_NO_WARNINGS
#endif

#include <hip/hip_runtime.h>
//#include ""

#include "kernel.cuh"

#define SHARED_SIZE_GO	(32 * 8)


///////////////////////////////////////////////////////////////////////////////////////////////////
// 
__global__ void GetOutput(const int nTrackX, double *pY, double *pSenseX, double *pSenseY, double *pSenseZ, double *pMagX, double *pMagY, double *pMagZ, const int nLayer, const int nHeadXMax, const int nHeadYMax, const int nMagXMax, const int nMagYMax, const int nOffset)
{
	__shared__ double dY[SHARED_SIZE_GO + 1];

	if (threadIdx.x < nHeadXMax)
	{
		dY[threadIdx.x] = 0.0f;

		const int nHeadXCentre = nHeadXMax / 2;
		const int nHeadYCentre = nHeadYMax / 2;

		int nXStart = nHeadXCentre - nTrackX;
		if (nXStart < 0)
			nXStart = 0;

		int nXStop = nMagXMax + nHeadXCentre - nTrackX;
		if (nXStop > nHeadXMax)
			nXStop = nHeadXMax;

		int nYStart = nHeadYCentre - blockIdx.x;
		if (nYStart < 0)
			nYStart = 0;

		int nYStop = nMagYMax + nHeadYCentre - blockIdx.x;
		if (nYStop > nHeadYMax)
			nYStop = nHeadYMax;

		int i, j;
		const int nSXY = (nHeadXMax * nHeadYMax * nLayer) + nXStart + threadIdx.x;
		const int nMXY = (nMagXMax * nMagYMax * nLayer) + nTrackX + nXStart - nHeadXCentre + threadIdx.x;

		for (i = nYStart; i < nYStop; i++)
		{
			int nIndexS = nSXY + (nHeadXMax * i);
			int nIndexM = nMXY + (nMagXMax * (blockIdx.x + i - nHeadYCentre));

			for (j = (nXStart + threadIdx.x); j < nXStop; j += blockDim.x)
			{
				dY[threadIdx.x] += ((pSenseX[nIndexS] * pMagX[nIndexM]) + (pSenseY[nIndexS] * pMagY[nIndexM]) + (pSenseZ[nIndexS] * pMagZ[nIndexM]));

				nIndexM += blockDim.x;
				nIndexS += blockDim.x;
			}
		}

		// Warp���Œl���܂Ƃ߂�
		for (i = (SHARED_SIZE_GO >> 1); i > 0; i >>= 1)
		{
			__syncthreads();

			if (i > threadIdx.x)
				dY[threadIdx.x] += dY[threadIdx.x + i];
		}

		if (0 == threadIdx.x)
			pY[nOffset + blockIdx.x] = dY[0];
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////
// 
void DeviceInit()
{
	hipSetDevice(0);
	DeviceReset();
}

///////////////////////////////////////////////////////////////////////////////////////////////////
// 
void DeviceReset()
{
	hipDeviceReset();
}

///////////////////////////////////////////////////////////////////////////////////////////////////
// 
int GetMagMemory(const int nPoint, const int nCrossTrack, const int nLayer, const int nSizeMag, ReciprocityMemory &RM)
{
	RM.nPoint = nPoint;
	RM.nCrossTrack = nCrossTrack;
	RM.nLayerMax = nLayer;

	// �Đݒ�͋������������m�ۂ�1�񂾂��B
	if (false != RM.bLoadMag)
		return 0;

	RM.nSize = (RM.nPoint + 50) * (RM.nCrossTrack + 20) * (RM.nLayerMax + 10);

	RM.h_Y = (double*)malloc(RM.nSize * sizeof(double));

	hipError_t error = hipSuccess;
	error = hipMalloc((void**)&(RM.d_MagX), (nSizeMag * 2 * sizeof(double)));
	error = hipMalloc((void**)&(RM.d_MagY), (nSizeMag * 2 * sizeof(double)));
	error = hipMalloc((void**)&(RM.d_MagZ), (nSizeMag * 2 * sizeof(double)));
	error = hipMalloc((void**)&(RM.d_Y), (RM.nSize * sizeof(double)));

	RM.bLoadMag = true;

	return error;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
// 
int ReleaseMemoryMag(ReciprocityMemory &RM)
{
	hipError_t error = hipSuccess;
	error = hipFree(RM.d_MagX);
	error = hipFree(RM.d_MagY);
	error = hipFree(RM.d_MagZ);
	error = hipFree(RM.d_Y);

	free(RM.h_Y);

	RM.Clear();

	return error;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
// 
int ReleaseMemoryHead(HeadMemory &HM)
{
	hipError_t error = hipSuccess;
	error = hipFree(HM.d_HeadX);
	error = hipFree(HM.d_HeadY);
	error = hipFree(HM.d_HeadZ);

	HM.Clear();

	return error;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
// 
int ReciprocitySetHead(HeadMemory &HM, int nSizeSense, sense *sSense)
{
	if (false != HM.bLoadHead)
		return 0;

	hipError_t error = hipSuccess;
	error = hipMalloc((void**)&(HM.d_HeadX), (nSizeSense * sizeof(double)));
	error = hipMalloc((void**)&(HM.d_HeadY), (nSizeSense * sizeof(double)));
	error = hipMalloc((void**)&(HM.d_HeadZ), (nSizeSense * sizeof(double)));
	error = hipMemcpy(HM.d_HeadX, sSense->dX, (nSizeSense * sizeof(double)), hipMemcpyHostToDevice);
	error = hipMemcpy(HM.d_HeadY, sSense->dY, (nSizeSense * sizeof(double)), hipMemcpyHostToDevice);
	error = hipMemcpy(HM.d_HeadZ, sSense->dZ, (nSizeSense * sizeof(double)), hipMemcpyHostToDevice);

	HM.bLoadHead = true;

	return error;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
// 
int ReciprocityGetOutput(ReciprocityMemory &RM, HeadMemory &HM, int nSizeMag, sense *sMag, int nLayerMin, const int nCrossTrackOffset, const int nLayerOffset)
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
		return error;

	error = hipMemcpy(RM.d_MagX, sMag->dX, (nSizeMag * sizeof(double)), hipMemcpyHostToDevice);
	error = hipMemcpy(RM.d_MagY, sMag->dY, (nSizeMag * sizeof(double)), hipMemcpyHostToDevice);
	error = hipMemcpy(RM.d_MagZ, sMag->dZ, (nSizeMag * sizeof(double)), hipMemcpyHostToDevice);
//	error = hipMemset(RM.d_Y, 0, (RM.nPoint * RM.nCrossTrack * RM.nLayerMax * sizeof(double)));
//	error = hipMemset(RM.d_Y, 0, (RM.nSize * sizeof(double)));

	dim3 Grid(1, 1, 1);
	dim3 Block(1, 1, 1);

	Grid.x = RM.nPoint;
	Grid.y = 1;
	Block.x = SHARED_SIZE_GO;
	Block.y = 1;

	for (int i = 0; i < RM.nCrossTrack; i++)
	{
		for (int j = nLayerMin; j <= RM.nLayerMax; j++)
			GetOutput<<<Grid, Block>>>(i, RM.d_Y, HM.d_HeadX, HM.d_HeadY, HM.d_HeadZ, RM.d_MagX, RM.d_MagY, RM.d_MagZ, j, HM.nHeadX, HM.nHeadY, sMag->nXMax, sMag->nYMax, ((nCrossTrackOffset * i) + (nLayerOffset * j)));
	}

//	error = hipMemcpy(RM.h_Y, RM.d_Y, (RM.nPoint * RM.nCrossTrack * RM.nLayerMax * sizeof(double)), hipMemcpyDeviceToHost);
	error = hipMemcpy(RM.h_Y, RM.d_Y, (RM.nSize * sizeof(double)), hipMemcpyDeviceToHost);

	return error;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
// 
bool CheckGPU()
{
	hipDeviceProp_t Property;
	memset(&Property, 0, sizeof(hipDeviceProp_t));

	// �g�p�ł��鐔���擾���܂��B
	int nCount = 0;
	hipError_t error = hipGetDeviceCount(&nCount);
	if (hipSuccess != error)
		return false;

	if (0 == nCount)
		return false;

	// �g���̂����߂܂��B
	for (int i = 0; i < nCount; i++)
	{
		error = hipSetDevice(i);
		if (hipSuccess != error)
			continue;

		error = hipGetDeviceProperties(&Property, i);
		if (hipSuccess != error)
			continue;

		// CC��7��肵���Ȃ�g�p�ł��Ȃ����܂��B
		if (6 > Property.major)
			continue;

		break;
	}

	return true;
}


#include "hip/hip_runtime.h"
/** 
problem: three rate equations:
    dm1/dt = m3*f2 - m2*f3 + g1 - m*g*m1
    dm2/dt = m1*f3 - m3*f1 + g2 - m*g*m2
    dm3/dt = m2*f1 - m1*f2 + g3 - m*g*m3
on the interval from t = 0.0 to t = 4.e10, with 
This program solves the problem with the BDF method
*/

#include <cvode/cvode.h> /* prototypes for CVODE fcts., consts.           */
#include <nvector/nvector_cuda.h> /* access to cuda N_Vector                       */
#include <stdio.h>
#include <stdlib.h>
#include <sundials/sundials_types.h> /* defs. of sunrealtype, int                        */
#include <sunlinsol/sunlinsol_spgmr.h>
#include <sunnonlinsol/sunnonlinsol_newton.h>
#include <math.h>

// constant memory

/* Problem Constants */ 
#define GROUPSIZE 3               /* number of equations per group */
#define indexbound 2
#define ONE 1
#define TWO 2
#define RTOL      SUN_RCONST(1.0e-5) /* scalar relative tolerance            */
#define ATOL1     SUN_RCONST(1.0e-5) /* vector absolute tolerance components */
#define ATOL2     SUN_RCONST(1.0e-5)
#define ATOL3     SUN_RCONST(1.0e-5)
#define T0        SUN_RCONST(0.0)  /* initial time           */
#define T1        SUN_RCONST(0.1)  /* first output time      */
#define DT    ((T1 - T0) / NOUT)
// #define NOUT      120             /* number of output times */

#define ZERO SUN_RCONST(0.0)

// constant memory
__constant__ float msk[3]={0.0f,0.0f,1.0f};
__constant__ float nsk[3]={1.0f,0.0f,0.0f};
__constant__ float chk=1.0f;
__constant__ float che =0.0f;
__constant__ float alpha=0.02f; // 0.0f
__constant__ float chg = 1.0f; 
__constant__ float cha = 1.5f; //0.2
__constant__ float chb = 0.0f;

/* user data structure for parallel*/
typedef struct
{
    // int ngroups; // number of groups
    int nx, ny;
    int neq; // number of equations
    sunrealtype *d_h;
    sunrealtype *d_mh;
} UserData;

/*
 *-------------------------------
 * Functions called by the solver
 *-------------------------------
 */

/* Right hand side function evaluation kernel. */
__global__ static void f_kernel(
  const sunrealtype* y, 
  sunrealtype* yd, 
  sunrealtype* h,
  sunrealtype* mh,
  int nx, 
  int ny,
  bool is_red_phase)
{
    // compute 2D thread coordinates
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= nx || iy >= ny) return;

    // checkerboard partition: red if (i+j) % 2 == 0
    bool is_red = ((ix+iy)&1) == 0;
    if (is_red != is_red_phase) return;

    // linear group index and base pointer for 3 components
    int gid = iy * nx + ix;
    int base_idx = GROUPSIZE * gid;

    // neighbor group indices
    int ix_l = (ix > 0) ? ix - 1 : ix;
    int ix_r = (ix < nx-1) ? ix + 1 : ix;
    int iy_u = (iy > 0) ? iy - 1 : iy;
    int iy_d = (iy < ny-1) ? iy + 1 : iy;

    int base_l = GROUPSIZE * (iy * nx + ix_l);
    int base_r = GROUPSIZE * (iy * nx + ix_r);
    int base_u = GROUPSIZE * (iy_u * nx + ix);
    int base_d = GROUPSIZE * (iy_d * nx + ix);

    // compute h vector for each component
    for (int c = 0; c < GROUPSIZE; ++c) {
        h[base_idx + c] =
            che * (y[base_l + c] + y[base_r + c] + y[base_u + c] + y[base_d + c])
          + msk[c] * (chk * y[base_idx + 2] + cha)
          + nsk[c] * (y[base_r + c] + y[base_l + c]) * chb;
    }
    __syncthreads();
    // Dot product m*h for this group
    sunrealtype dot = 
        y[base_idx + 0] * h[base_idx + 0] +
        y[base_idx + 1] * h[base_idx + 1] +
        y[base_idx + 2] * h[base_idx + 2];
    mh[base_idx + 0] = dot;
    mh[base_idx + 1] = dot;
    mh[base_idx + 2] = dot;
    __syncthreads();
    yd[base_idx + 0] = chg * (y[base_idx + 2]*h[base_idx + 1] - y[base_idx + 1]*h[base_idx + 2])
                      + alpha * (h[base_idx + 0] - dot * y[base_idx + 0]);
    yd[base_idx + 1] = chg * (y[base_idx + 0]*h[base_idx + 2]
                             - y[base_idx + 2]*h[base_idx + 0])
                     + alpha * (h[base_idx + 1] - dot * y[base_idx + 1]);
    yd[base_idx + 2] = chg * (y[base_idx + 1]*h[base_idx + 0]
                             - y[base_idx + 0]*h[base_idx + 1])
                     + alpha * (h[base_idx + 2] - dot * y[base_idx + 2]);
}

/* Right hand side function. This just launches the CUDA kernel
   to do the actual computation. At the very least, doing this
   saves moving the vector data in y and ydot to/from the device
   every evaluation of f. */

static int f(sunrealtype t, N_Vector y, N_Vector ydot, void* user_data)
{
    UserData* udata;
    sunrealtype *ydata, *ydotdata;

    udata    = (UserData*)user_data;
    ydata    = N_VGetDeviceArrayPointer_Cuda(y);
    ydotdata = N_VGetDeviceArrayPointer_Cuda(ydot);

    int nx = udata->nx, ny = udata->ny;
    dim3 block(16, 16);
    dim3 grid((nx + block.x - 1)/block.x, (ny + block.y - 1)/block.y);

    // Red phase
    f_kernel<<<grid, block>>>(ydata, ydotdata,
                                 udata->d_h, udata->d_mh,
                                 nx, ny, true);
    hipDeviceSynchronize();

    // Blue phase
    f_kernel<<<grid, block>>>(ydata, ydotdata,
                                 udata->d_h, udata->d_mh,
                                 nx, ny, false);
    hipDeviceSynchronize();
    
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, ">>> ERROR in f: hipGetLastError returned %s\n",
                hipGetErrorName(cuerr));
        return (-1);
    }

    return (0);
}

/*
 *-------------------------------
 * Private helper functions
 *-------------------------------
 */
static void PrintOutput(sunrealtype t, sunrealtype y1, sunrealtype y2,
                        sunrealtype y3)
{
    #if defined(SUNDIALS_EXTENDED_PRECISION)
      printf("At t = %0.4Le      y =%14.6Le  %14.6Le  %14.6Le\n", t, y1, y2, y3);
    #elif defined(SUNDIALS_DOUBLE_PRECISION)
      printf("At t = %0.4e      y =%14.6e  %14.6e  %14.6e\n", t, y1, y2, y3);
    #else
      printf("At t = %0.4e      y =%14.6e  %14.6e  %14.6e\n", t, y1, y2, y3);
    #endif

  return;
}

/*
 * Get and print some final statistics
 */
static void PrintFinalStats(void* cvode_mem, SUNLinearSolver LS)
{
    long int nst, nfe, nsetups, nni, ncfn, netf, nge;

    CVodeGetNumSteps(cvode_mem, &nst);
    CVodeGetNumRhsEvals(cvode_mem, &nfe);
    CVodeGetNumLinSolvSetups(cvode_mem, &nsetups);
    CVodeGetNumErrTestFails(cvode_mem, &netf);
    CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
    CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
    CVodeGetNumGEvals(cvode_mem, &nge);

    printf("\nFinal Statistics:\n");
    printf("nst = %-6ld nfe  = %-6ld nsetups = %-6ld", nst, nfe,
          nsetups);
    printf("nni = %-6ld ncfn = %-6ld netf = %-6ld    nge = %ld\n", nni, ncfn,
          netf, nge);
}

/*
 *-------------------------------
 * Main Program
 *-------------------------------
 */
int main(int argc, char* argv[])
{
    SUNContext sunctx; // SUNDIALS context
    sunrealtype *ydata, *abstol_data; // Host-side pointers to solution and tolerance data
    sunrealtype t;
    sunrealtype tout;
    N_Vector y, abstol; // SUNDIALS vector structures for solution and absolute tolerance
    SUNLinearSolver LS; // Linear solver object (cuSolverSp QR)
    SUNNonlinearSolver NLS;
    void* cvode_mem; // CVODE integrator memory
    int retval, iout; // return status and output counter
    int neq, ngroups, groupj;// Problem size: number of equations, groups, and loop index
    UserData udata;

    /* Parse command-line to get number of groups */
    int nx = 128, ny = 128; 
    neq     = nx * ny * 3;

    /* Fill user data */
    udata.nx  = nx;
    udata.ny  = ny;
    udata.neq     = neq;
    hipMalloc(&udata.d_h,  neq * sizeof(sunrealtype));
    hipMalloc(&udata.d_mh, neq * sizeof(sunrealtype));

    /* Create SUNDIALS context */
    SUNContext_Create(SUN_COMM_NULL, &sunctx);

    /* Allocate CUDA vectors for solution and tolerances */
    y     = N_VNew_Cuda(neq, sunctx);
    abstol= N_VNew_Cuda(neq, sunctx);
    // get host pointers
    ydata       = N_VGetHostArrayPointer_Cuda(y);
    abstol_data = N_VGetHostArrayPointer_Cuda(abstol);

    /* Initialize y and abstol on host then copy to device */
    for (int j = 0; j < ny; ++j) {
      for (int i = 0; i < nx; ++i) {
        int idx = 3 * (j * nx + i);
        if (j < ny / 2) {
          ydata[idx + 0] = 0.0;
          ydata[idx + 1] = 0.0175;
          ydata[idx + 2] = 0.998;
        } else {
          ydata[idx + 0] = 0.0;
          ydata[idx + 1] = 0.0175;
          ydata[idx + 2] = -0.998;
        }

        abstol_data[idx + 0] = ATOL1;
        abstol_data[idx + 1] = ATOL2;
        abstol_data[idx + 2] = ATOL3;
      }
    }
    N_VCopyToDevice_Cuda(y);
    N_VCopyToDevice_Cuda(abstol);

    /* Create and initialize CVODE solver memory */
    cvode_mem = CVodeCreate(CV_BDF, sunctx);
    CVodeInit(cvode_mem, f, T0, y);
    CVodeSetUserData(cvode_mem, &udata);
    CVodeSVtolerances(cvode_mem, RTOL, abstol);

    /* Matrix-free GMRES linear solver (no Jacobian needed) */
    NLS = SUNNonlinSol_Newton(y, sunctx);
    CVodeSetNonlinearSolver(cvode_mem, NLS);
    LS = SUNLinSol_SPGMR(y, SUN_PREC_NONE, 0, sunctx);
    CVodeSetLinearSolver(cvode_mem, LS, NULL);

    /* Print header */
    printf("\nGroup of independent 3-species kinetics problems\n\n");
    printf("number of groups = %d\n\n", nx, ny, nx * ny);

    /* Time-stepping loop */
    float ttotal=500.0f;
    iout = T0;
    tout = T1;
    int NOUT=ttotal/T1;
    while (iout < NOUT) {
      // &t cvode实际走到的地方
      retval = CVode(cvode_mem, tout, y, &t, CV_NORMAL);
      // copy solution back to host and print all groups
      if (retval == CV_SUCCESS) {
        iout++;
        tout += T1; // T0 + iout*T1
      }else {
        fprintf(stderr, "CVode error at output %d: retval = %d\n", iout, retval);
        break;
      }
      // printf("%f\n",tout);
      N_VCopyFromDevice_Cuda(y);
      ydata = N_VGetHostArrayPointer_Cuda(y);
      int mid = (ny/2) * nx + (nx/2);
      printf("t = %.2f, m = [%g %g %g]\n", t,
              ydata[3*mid+0], ydata[3*mid+1], ydata[3*mid+2]);
    }

    /* Print final statistics */
    PrintFinalStats(cvode_mem, LS);

    /* Clean up */
    hipFree(udata.d_h);
    hipFree(udata.d_mh);
    N_VDestroy(y);
    N_VDestroy(abstol);
    CVodeFree(&cvode_mem);
    SUNLinSolFree(LS);
    SUNNonlinSolFree(NLS);
    SUNContext_Free(&sunctx);

    return 0;
}

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define CUDA_CALL(err) \
    if ((err) != hipSuccess) { \
        fprintf(stderr, "CUDA error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    }

#define CUFFT_CALL(err) \
    if ((err) != HIPFFT_SUCCESS) { \
        fprintf(stderr, "CUFFT error %d at %s:%d\n", (int)(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    }

typedef hipfftComplex Complex;

const int NX = 128, NY = 128;
const int KX =  64, KY =  64;
const int PX = NX + KX - 1;
const int PY = NY + KY - 1;
const int SIZE = PX * PY;

__global__ void complexMul(Complex *H, const Complex *A, const Complex *B, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float ar = A[idx].x, ai = A[idx].y;
        float br = B[idx].x, bi = B[idx].y;
        H[idx].x = ar * br - ai * bi;
        H[idx].y = ar * bi + ai * br;
    }
}

__global__ void complexMulAdd(Complex *H, const Complex *A, const Complex *B, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float ar = A[idx].x, ai = A[idx].y;
        float br = B[idx].x, bi = B[idx].y;
        H[idx].x += ar * br - ai * bi;
        H[idx].y += ar * bi + ai * br;
    }
}

__global__ void normalize(Complex *data, int N, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx].x *= scale;
        data[idx].y *= scale;
    }
}

void padAndConvert(const float *in, Complex *out, int wx, int wy) {
    for (int i = 0; i < SIZE; ++i) out[i].x = out[i].y = 0.0f;
    for (int j = 0; j < wy; ++j)
        for (int i = 0; i < wx; ++i)
            out[i + j * PX].x = in[i + j * wx];
}

static bool approx(double a, double b, double tol_abs, double tol_rel=0.0) {
    double diff = std::fabs(a - b);
    return diff <= tol_abs || diff <= tol_rel * std::fabs(b);
}

static void require(bool cond, const char* what, int &fails) {
    if (!cond) { fprintf(stderr, "FAIL: %s\n", what); ++fails; }
    else       { printf("PASS: %s\n", what); }
}

int main() {
    const int MN = NX*NY, KN = KX*KY;
    float *h_Mx = (float*)malloc(MN * sizeof(float));
    float *h_My = (float*)malloc(MN * sizeof(float));
    float *h_Mz = (float*)malloc(MN * sizeof(float));
    float *h_D[9];
    for (int i = 0; i < 9; i++) h_D[i] = (float*)malloc(KN * sizeof(float));

    for (int j = 0; j < NY; j++)
        for (int i = 0; i < NX; i++) {
            int idx = i + j * NX;
            h_Mx[idx] = 1.0f; h_My[idx] = 0.0f; h_Mz[idx] = 0.0f;
        }
    for (int j = 0; j < KY; j++)
        for (int i = 0; i < KX; i++) {
            int idx = i + j * KX;
            h_D[0][idx] = 1.0f; h_D[1][idx] = 0.0f; h_D[2][idx] = 0.0f;
            h_D[3][idx] = 0.0f; h_D[4][idx] = 1.0f; h_D[5][idx] = 0.0f;
            h_D[6][idx] = 0.0f; h_D[7][idx] = 0.0f; h_D[8][idx] = 1.0f;
        }

    Complex *h_Mx_p = (Complex*)malloc(SIZE * sizeof(Complex));
    Complex *h_My_p = (Complex*)malloc(SIZE * sizeof(Complex));
    Complex *h_Mz_p = (Complex*)malloc(SIZE * sizeof(Complex));
    Complex *h_D_p[9];
    for (int i = 0; i < 9; i++) h_D_p[i] = (Complex*)malloc(SIZE * sizeof(Complex));

    padAndConvert(h_Mx, h_Mx_p, NX, NY);
    padAndConvert(h_My, h_My_p, NX, NY);
    padAndConvert(h_Mz, h_Mz_p, NX, NY);
    for (int i = 0; i < 9; i++) padAndConvert(h_D[i], h_D_p[i], KX, KY);

    Complex *d_Mx, *d_My, *d_Mz, *d_Hx, *d_Hy, *d_Hz;
    Complex *d_D[9];
    CUDA_CALL(hipMalloc(&d_Mx, SIZE * sizeof(Complex)));
    CUDA_CALL(hipMalloc(&d_My, SIZE * sizeof(Complex)));
    CUDA_CALL(hipMalloc(&d_Mz, SIZE * sizeof(Complex)));
    CUDA_CALL(hipMalloc(&d_Hx, SIZE * sizeof(Complex)));
    CUDA_CALL(hipMalloc(&d_Hy, SIZE * sizeof(Complex)));
    CUDA_CALL(hipMalloc(&d_Hz, SIZE * sizeof(Complex)));
    for (int i = 0; i < 9; i++) CUDA_CALL(hipMalloc(&d_D[i], SIZE * sizeof(Complex)));

    CUDA_CALL(hipMemcpy(d_Mx, h_Mx_p, SIZE * sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_My, h_My_p, SIZE * sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_Mz, h_Mz_p, SIZE * sizeof(Complex), hipMemcpyHostToDevice));
    for (int i = 0; i < 9; i++)
        CUDA_CALL(hipMemcpy(d_D[i], h_D_p[i], SIZE * sizeof(Complex), hipMemcpyHostToDevice));

    hipfftHandle plan;
    CUFFT_CALL(hipfftPlan2d(&plan, PY, PX, HIPFFT_C2C));

    for (int i = 0; i < 9; i++) CUFFT_CALL(hipfftExecC2C(plan, d_D[i], d_D[i], HIPFFT_FORWARD));
    CUFFT_CALL(hipfftExecC2C(plan, d_Mx, d_Mx, HIPFFT_FORWARD));
    CUFFT_CALL(hipfftExecC2C(plan, d_My, d_My, HIPFFT_FORWARD));
    CUFFT_CALL(hipfftExecC2C(plan, d_Mz, d_Mz, HIPFFT_FORWARD));

    CUDA_CALL(hipMemset(d_Hx, 0, SIZE * sizeof(Complex)));
    CUDA_CALL(hipMemset(d_Hy, 0, SIZE * sizeof(Complex)));
    CUDA_CALL(hipMemset(d_Hz, 0, SIZE * sizeof(Complex)));

    int thr = 256, blk = (SIZE + thr - 1) / thr;
    complexMul <<<blk,thr>>>(d_Hx, d_Mx, d_D[0], SIZE);
    complexMulAdd<<<blk,thr>>>(d_Hx, d_My, d_D[1], SIZE);
    complexMulAdd<<<blk,thr>>>(d_Hx, d_Mz, d_D[2], SIZE);
    complexMul <<<blk,thr>>>(d_Hy, d_Mx, d_D[3], SIZE);
    complexMulAdd<<<blk,thr>>>(d_Hy, d_My, d_D[4], SIZE);
    complexMulAdd<<<blk,thr>>>(d_Hy, d_Mz, d_D[5], SIZE);
    complexMul <<<blk,thr>>>(d_Hz, d_Mx, d_D[6], SIZE);
    complexMulAdd<<<blk,thr>>>(d_Hz, d_My, d_D[7], SIZE);
    complexMulAdd<<<blk,thr>>>(d_Hz, d_Mz, d_D[8], SIZE);

    CUFFT_CALL(hipfftExecC2C(plan, d_Hx, d_Hx, HIPFFT_BACKWARD));
    CUFFT_CALL(hipfftExecC2C(plan, d_Hy, d_Hy, HIPFFT_BACKWARD));
    CUFFT_CALL(hipfftExecC2C(plan, d_Hz, d_Hz, HIPFFT_BACKWARD));
    float scale = 1.0f / SIZE;
    normalize<<<blk,thr>>>(d_Hx, SIZE, scale);
    normalize<<<blk,thr>>>(d_Hy, SIZE, scale);
    normalize<<<blk,thr>>>(d_Hz, SIZE, scale);
    CUDA_CALL(hipDeviceSynchronize());

    Complex *h_Hx = (Complex*)malloc(SIZE * sizeof(Complex));
    Complex *h_Hy = (Complex*)malloc(SIZE * sizeof(Complex));
    Complex *h_Hz = (Complex*)malloc(SIZE * sizeof(Complex));
    CUDA_CALL(hipMemcpy(h_Hx, d_Hx, SIZE * sizeof(Complex), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_Hy, d_Hy, SIZE * sizeof(Complex), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_Hz, d_Hz, SIZE * sizeof(Complex), hipMemcpyDeviceToHost));

    // int cx = (PX/2) + (PY/2)*PX;
    // printf("Hx(中心) = %f\n", h_Hx[cx].x);
    // printf("Hy(中心) = %f\n", h_Hy[cx].x);
    // printf("Hz(中心) = %f\n", h_Hz[cx].x);

    // ----- Tests -----
    int fails = 0;
    const double EPS = 1e-3;

    // 1) Center/plateau value ≈ 64*64
    int cx = (PX/2) + (PY/2)*PX;
    double center = h_Hx[cx].x;
    printf("Hx(center) = %.6f\n", center);
    require(approx(center, 64.0*64.0, 1e-2, 1e-6), "Hx center ≈ 4096", fails);

    // 2) Imag parts small everywhere, and Hy/Hz near zero
    double max_real_Hx = 0, max_im_Hx=0, max_abs_Hy=0, max_abs_Hz=0;
    for (int i=0;i<SIZE;i++) {
        max_real_Hx = fmax(max_real_Hx, fabs((double)h_Hx[i].x));
        max_im_Hx = fmax(max_im_Hx, std::fabs(h_Hx[i].y));
        max_abs_Hy = fmax(max_abs_Hy, std::fabs(h_Hy[i].x));
        max_abs_Hz = fmax(max_abs_Hz, std::fabs(h_Hz[i].x));
    }
    double rel_im = (max_real_Hx > 0) ? max_im_Hx / max_real_Hx : 0.0;
    require(rel_im < 1e-6, "Imag(Hx) small (relative)", fails);

    // 3) Sum(Hx) ≈ 16384 * 4096
    long double sumHx = 0.0L;
    for (int i=0;i<SIZE;i++) sumHx += (long double)h_Hx[i].x;
    long double expectedSum = (long double)(NX*NY) * (long double)(KX*KY); // 67108864
    printf("sum(Hx) = %.0Lf (expected %.0Lf)\n", sumHx, expectedSum);
    require(fabsl(sumHx - expectedSum) <= 1e-2L * expectedSum, "Sum(Hx) conservation", fails);

    if (fails == 0) {
        printf("ALL TESTS PASSED\n");
    } else {
        printf("%d TEST(S) FAILED\n", fails);
    }

    CUFFT_CALL(hipfftDestroy(plan));
    hipFree(d_Mx); hipFree(d_My); hipFree(d_Mz);
    hipFree(d_Hx); hipFree(d_Hy); hipFree(d_Hz);
    for (int i = 0; i < 9; i++) hipFree(d_D[i]);
    free(h_Mx); free(h_My); free(h_Mz);
    free(h_Hx); free(h_Hy); free(h_Hz);
    free(h_Mx_p); free(h_My_p); free(h_Mz_p);
    for (int i = 0; i < 9; i++) { free(h_D[i]); free(h_D_p[i]); }
    return (fails==0) ? 0 : 1;
}

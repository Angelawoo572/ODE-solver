#include "hip/hip_runtime.h"
/** 
problem: three rate equations:
    dm1/dt = m3*f2 - m2*f3 + g1 - m*g*m1
    dm2/dt = m1*f3 - m3*f1 + g2 - m*g*m2
    dm3/dt = m2*f1 - m1*f2 + g3 - m*g*m3
on the interval from t = 0.0 to t = 4.e10, with initial
conditions: m1 = 1.0, m2 = 0.0, m3 = 0.0
This program solves the problem with the BDF method
*/

#include <cvode/cvode.h> /* prototypes for CVODE fcts., consts.           */
#include <nvector/nvector_cuda.h> /* access to cuda N_Vector                       */
#include <stdio.h>
#include <stdlib.h>
#include <sundials/sundials_types.h> /* defs. of sunrealtype, int                        */
#include <sunlinsol/sunlinsol_cusolversp_batchqr.h>
/**
CUDA linear solver (if using cusolver)
access to cuSolverSp batch QR SUNLinearSolver */
#include <sunmatrix/sunmatrix_cusparse.h> /* access to cusparse SUNMatrix  */

/* Problem Constants */
#define GROUPSIZE 3               /* number of equations per group */
/* 我们每个 block 是 3×3，所以每组非零数 nnzper = 9 */
const int nnzper = GROUPSIZE * GROUPSIZE;
#define Y1        SUN_RCONST(1.0) /* initial y components */
#define Y2        SUN_RCONST(0.0)
#define Y3        SUN_RCONST(0.0)
#define RTOL      SUN_RCONST(1.0e-4) /* scalar relative tolerance            */
#define ATOL1     SUN_RCONST(1.0e-8) /* vector absolute tolerance components */
#define ATOL2     SUN_RCONST(1.0e-14)
#define ATOL3     SUN_RCONST(1.0e-6)
#define T0        SUN_RCONST(0.0)  /* initial time           */
#define T1        SUN_RCONST(10.0)  /* first output time      */
#define TMULT     SUN_RCONST(10.0) /* output time factor     */
#define NOUT      12               /* number of output times */

#define ZERO SUN_RCONST(0.0)

/* Functions Called by the Solver */

static int f(sunrealtype t, N_Vector y, N_Vector yd, void* user_data);

static int Jac(sunrealtype t, N_Vector y, N_Vector fy, SUNMatrix J,
               void* user_data, N_Vector tmp1, N_Vector tmp2, N_Vector tmp3);

/* Private function to initialize the Jacobian sparsity pattern */
static int JacInit(SUNMatrix J);

/* Private function to output results */

static void PrintOutput(sunrealtype t, sunrealtype y1, sunrealtype y2,
                        sunrealtype y3);

/* Private function to print final statistics */

static void PrintFinalStats(void* cvode_mem, SUNLinearSolver LS);


/* user data structure for parallel*/
typedef struct
{
  int ngroups; // number of groups
  int neq; // number of equations
  sunrealtype fi, fj, fk;
  sunrealtype g1, g2,g3;
  sunrealtype g,m;
} UserData;

/*
 *-------------------------------
 * Functions called by the solver
 *-------------------------------
 */

/* Right hand side function evaluation kernel. */
__global__ static void f_kernel(sunrealtype t, sunrealtype* y,
                                sunrealtype* yd, int neq ) 
{
  sunrealtype fi, fj, fk;
  sunrealtype gi, gj, gk;
  sunrealtype gm;

  __constant__ float he,h

  int i      = threadIdx.x;
  int j      = blockDim.x +threadIdx.x;
  int k      = 2*blockDim.x +threadIdx.x;


  if (i < neq -1 && i > 0 )
  {
    fi = he*(y[i+1]+y[i-1]);
    fj = he*(y[j+1]+y[i-1]);
    fk = he*(y[k+1]+y[i-1])+hk*y[k]+hap;

    gi = ap*fi;
    gj = ap*fj;
    gk = ap*fk;

    mg=y[i]*gi+y[j]*gj+y[k]*gk;

    yd[i] = y[k]*fj - y[j]*fk + gi - mg*y[i];
    yd[j] = y[i]*fk - y[k]*fi + gj - mg*y[j];
    yd[k] = y[j]*fi - y[i]*fj + gk - mg*y[k];
  }
  else
  {
	  yd[i]=0;
	  yd[j]=0;
	  yd[k]=0;
  }
}

/* Right hand side function. This just launches the CUDA kernel
   to do the actual computation. At the very least, doing this
   saves moving the vector data in y and ydot to/from the device
   every evaluation of f. */
static int f(sunrealtype t, N_Vector y, N_Vector ydot, void* user_data)
{
    UserData* udata;
    sunrealtype *y, *yd;

    u= (UserData*)user_data;
    y= N_VGetDeviceArrayPointer_Cuda(y);
    yd= N_VGetDeviceArrayPointer_Cuda(ydot);

    unsigned block_size = 32;
    // total threads = grid_size * block_size
    // grid_size is ceil - (a+b-1)/b
    unsigned grid_size  = (udata->neq + block_size - 1) / block_size;
    f_kernel<<<grid_size, block_size>>>(t, y, yd, u->neq,u->ngroups);

    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, ">>> ERROR in f: hipGetLastError returned %s\n",
                hipGetErrorName(cuerr));
        return (-1);
    }

    return (0);
}

/*
 *-------------------------------
 * Private helper functions
 *-------------------------------
 */
static void PrintOutput(sunrealtype t, sunrealtype y1, sunrealtype y2,
                        sunrealtype y3)
{
#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf("At t = %0.4Le      y =%14.6Le  %14.6Le  %14.6Le\n", t, y1, y2, y3);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf("At t = %0.4e      y =%14.6e  %14.6e  %14.6e\n", t, y1, y2, y3);
#else
  printf("At t = %0.4e      y =%14.6e  %14.6e  %14.6e\n", t, y1, y2, y3);
#endif

  return;
}

/*
 * Get and print some final statistics
 */
static void PrintFinalStats(void* cvode_mem, SUNLinearSolver LS)
{
  long int nst, nfe, nsetups, nni, ncfn, netf, nge;

  CVodeGetNumSteps(cvode_mem, &nst);
  CVodeGetNumRhsEvals(cvode_mem, &nfe);
  CVodeGetNumLinSolvSetups(cvode_mem, &nsetups);
  CVodeGetNumErrTestFails(cvode_mem, &netf);
  CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
  CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
  CVodeGetNumGEvals(cvode_mem, &nge);


  printf("\nFinal Statistics:\n");
  printf("nst = %-6ld nfe  = %-6ld nsetups = %-6ld", nst, nfe,
         nsetups);
  printf("nni = %-6ld ncfn = %-6ld netf = %-6ld    nge = %ld\n", nni, ncfn,
         netf, nge);
}

/*
 * Jacobian initialization routine. This sets the sparisty pattern of
 * the blocks of the Jacobian J(t,y) = df/dy. This is performed on the CPU,
 * and only occurs at the beginning of the simulation.
 */

static int JacInit(SUNMatrix J)
{
  int    rowptrs[GROUPSIZE+1];
  int    colvals[nnzper];

  /* 全置零 */
  SUNMatZero(J);

  /* compressed sparse row 的 rowptrs */
  for (int i = 0; i <= GROUPSIZE; i++)
    rowptrs[i] = i * GROUPSIZE;

  /* 每行的列索引 0,1,2 */
  for (int i = 0; i < nnzper; i++)
    colvals[i] = i % GROUPSIZE;

  /* copy rowptrs, colvals to the device */
  SUNMatrix_cuSparse_CopyToDevice(J, NULL, rowptrs, colvals);
  hipDeviceSynchronize();

  return (0);
}

/* Jacobian evaluation GPU kernel */
__global__ static void j_kernel(int ngroups,
                                sunrealtype f1, sunrealtype f2, sunrealtype f3,
                                sunrealtype g1, sunrealtype g2, sunrealtype g3,
                                sunrealtype m,  sunrealtype g,
                                sunrealtype* ydata,
                                sunrealtype* Jdata)
{
  int groupj;

  for (groupj = blockIdx.x * blockDim.x + threadIdx.x; groupj < ngroups;
       groupj += blockDim.x * gridDim.x)
  {

    /* first row of block: ∂f1/∂m1, ∂f1/∂m2, ∂f1/∂m3 */
    Jdata[nnzper * groupj + 0] = - m * g;
    Jdata[nnzper * groupj + 1] = - f3;
    Jdata[nnzper * groupj + 2] =   f2;

    /* second row of block: ∂f2/∂m1, ∂f2/∂m2, ∂f2/∂m3 */
    Jdata[nnzper * groupj + 3] =   f3;
    Jdata[nnzper * groupj + 4] = - m * g;
    Jdata[nnzper * groupj + 5] = - f1;

    /* third row of block: ∂f3/∂m1, ∂f3/∂m2, ∂f3/∂m3 */
    Jdata[nnzper * groupj + 6] = - f2;
    Jdata[nnzper * groupj + 7] =   f1;
    Jdata[nnzper * groupj + 8] = - m * g;
  }
}

/*
 * Jacobian routine. Compute J(t,y) = df/dy.
 * This is done on the GPU.
 */

static int Jac(sunrealtype t, N_Vector y, N_Vector fy, SUNMatrix J,
               void* user_data, N_Vector tmp1, N_Vector tmp2, N_Vector tmp3)
{
  UserData* udata = (UserData*)user_data;
  sunrealtype *Jdata, *ydata;
  unsigned block_size, grid_size;

  Jdata  = SUNMatrix_cuSparse_Data(J);
  ydata  = N_VGetDeviceArrayPointer_Cuda(y);

  block_size = 32;
  grid_size  = (udata->neq + block_size - 1) / block_size;
  j_kernel<<<grid_size,block_size>>>(udata->ngroups,
                                     udata->f1, udata->f2, udata->f3,
                                     udata->g1, udata->g2, udata->g3,
                                     udata->m,  udata->g,
                                     ydata, Jdata);

  hipDeviceSynchronize();
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess)
  {
    fprintf(stderr, ">>> ERROR in Jac: hipGetLastError returned %s\n",
            hipGetErrorName(cuerr));
    return (-1);
  }

  return (0);
}


/*
 *-------------------------------
 * Main Program
 *-------------------------------
 */
int main(int argc, char* argv[])
{
  SUNContext sunctx; // SUNDIALS context
  sunrealtype reltol, t, tout; // Solver tolerances and time variables
  sunrealtype *ydata, *abstol_data; // Host-side pointers to solution and tolerance data
  N_Vector y, abstol; // SUNDIALS vector structures for solution and absolute tolerance
  SUNMatrix A;
  SUNLinearSolver LS; // Linear solver object (cuSolverSp QR)
  void* cvode_mem; // CVODE integrator memory
  int retval, iout; // return status and output counter
  int neq, ngroups, groupj;// Problem size: number of equations, groups, and loop index
  UserData udata;
  hipsparseHandle_t cusp_handle;
  hipsolverSpHandle_t cusol_handle;

  y = abstol = NULL;// Initialize all pointers to NULL to ensure safe cleanup
  A = NULL;
  LS = NULL;  // Initialize linear solver pointer
  cvode_mem = NULL;  // Initialize CVODE memory


  /* Parse command line arguments */
  if (argc > 1) { ngroups = atoi(argv[1]); }
  else { ngroups = 100; }
  neq = ngroups * GROUPSIZE;

  udata.ngroups = ngroups;
  udata.neq     = neq;

  udata.f1 = 1.0;
  udata.f2 = 2.0;
  udata.f3 = 3.0;
  udata.g1 = 0.1;
  udata.g2 = 0.2;
  udata.g3 = 0.3;
  udata.g = 0.01;
  udata.m = 1.5;

  /* Initialize cuSOLVER and cuSPARSE handles */
  hipsparseCreate(&cusp_handle);
  hipsolverSpCreate(&cusol_handle);

  /* Create the SUNDIALS context */
  SUNContext_Create(SUN_COMM_NULL, &sunctx);

  /* Create CUDA vector of length neq for I.C. and abstol */
  y = N_VNew_Cuda(neq, sunctx);
  abstol = N_VNew_Cuda(neq, sunctx);

  ydata       = N_VGetHostArrayPointer_Cuda(y);
  abstol_data = N_VGetHostArrayPointer_Cuda(abstol);

  /* Initialize y */
  for (groupj = 0; groupj < neq; groupj += GROUPSIZE)
  {
    ydata[groupj]     = Y1;
    ydata[groupj + 1] = Y2;
    ydata[groupj + 2] = Y3;
  }
  N_VCopyToDevice_Cuda(y);

  /* Set the scalar relative tolerance */
  reltol = RTOL;

  /* Set the vector absolute tolerance */
  for (groupj = 0; groupj < neq; groupj += GROUPSIZE)
  {
    abstol_data[groupj]     = ATOL1;
    abstol_data[groupj + 1] = ATOL2;
    abstol_data[groupj + 2] = ATOL3;
  }
  N_VCopyToDevice_Cuda(abstol);

  /* Call CVodeCreate to create the solver memory and specify the
   * Backward Differentiation Formula */
  cvode_mem = CVodeCreate(CV_BDF, sunctx);

  /* Call CVodeInit to initialize the integrator memory and specify the
   * user's right hand side function in y'=f(t,y), the initial time T0, and
   * the initial dependent variable vector y. */
  CVodeInit(cvode_mem, f, T0, y);

  /* Call CVodeSetUserData to attach the user data structure */
  CVodeSetUserData(cvode_mem, &udata);

  /* Call CVodeSVtolerances to specify the scalar relative tolerance
   * and vector absolute tolerances */
  CVodeSVtolerances(cvode_mem, reltol, abstol);

  A = SUNMatrix_cuSparse_NewBlockCSR(ngroups, GROUPSIZE, GROUPSIZE,
                                     GROUPSIZE * GROUPSIZE, cusp_handle, sunctx);

  /* Set the sparsity pattern to be fixed so that the row pointers
   * and column indices are not zeroed out by SUNMatZero */
  SUNMatrix_cuSparse_SetFixedPattern(A, 1);
  /* Initialiize the Jacobian with its fixed sparsity pattern */
  JacInit(A);
  /* Create the SUNLinearSolver object for use by CVode */
  LS = SUNLinSol_cuSolverSp_batchQR(y, A, cusol_handle, sunctx);

  CVodeSetLinearSolver(cvode_mem, LS, A);

  /* Set the user-supplied Jacobian routine Jac */
  CVodeSetJacFn(cvode_mem, Jac);

  /* In loop, call CVode, print results, and test for error.
     Break out of loop when NOUT preset output times have been reached.  */
  printf(" \nGroup of independent 3-species kinetics problems\n\n");
  printf("number of groups = %d\n\n", ngroups);

  iout = 0;
  tout = T1;
  while (1)
  {
    retval = CVode(cvode_mem, tout, y, &t, CV_NORMAL);

    N_VCopyFromDevice_Cuda(y);
    for (groupj = 0; groupj < ngroups; groupj += 10)
    {
      printf("group %d: ", groupj);
      PrintOutput(t, ydata[GROUPSIZE * groupj], ydata[1 + GROUPSIZE * groupj],
                  ydata[2 + GROUPSIZE * groupj]);
    }
    if (retval == CV_SUCCESS)
    {
      iout++;
      tout *= TMULT;
    }

    if (iout == NOUT) { break; }
  }

  /* Print some final statistics */
  PrintFinalStats(cvode_mem, LS);

  /* Free y and abstol vectors */
  N_VDestroy(y);
  N_VDestroy(abstol);

  /* Free integrator memory */
  CVodeFree(&cvode_mem);

  /* Free the linear solver memory */
  SUNLinSolFree(LS);

  /* Free the matrix memory */
  SUNMatDestroy(A);

  SUNContext_Free(&sunctx);

  /* Destroy the cuSOLVER and cuSPARSE handles */
  hipsparseDestroy(cusp_handle);
  hipsolverSpDestroy(cusol_handle);

  return (0);
}

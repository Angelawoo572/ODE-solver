
#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>

using namespace std;


__global__ void kernel_call(int N, double *in, double* out)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  
  for (int i = id; i < N; i += blockDim.x * gridDim.x)
    out[i] = in[i];
}


int main(){

    double *host_in, *host_out;
    double *dev_in, *dev_out;

    size_t N = 18874368; 
 
		
    //create buffer on host	
    host_in = (double*) malloc(N* sizeof(double));
    host_out = (double*) malloc(N * sizeof(double));


    //create buffer on device
    hipError_t err = hipMalloc(&dev_in, N*sizeof(double));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }


    err = hipMalloc(&dev_out, N*sizeof(double));
    if (err != hipSuccess){
       cout<<"Dev Memory not allocated"<<endl;
       exit(-1);
    }



    for (int i = 1; i < 128; i <<= 1)
{
 
   cout<<i<<" "<<N/i  <<" ";
    
    //using OpenMP to perform timing on the host   
    double st = omp_get_wtime();
    hipMemcpy(dev_in, host_in, N * sizeof(double), hipMemcpyHostToDevice);
    double et = omp_get_wtime();
 
    cout<<"Copy time: "<<(et-st)*1000<<"ms ";     


    //create GPU timing events for timing the GPU
    hipEvent_t st2, et2;
    hipEventCreate(&st2);
    hipEventCreate(&et2);        
     
    hipEventRecord(st2);
    kernel_call<<<1, 32>>>(N, dev_in, dev_out);
    hipEventRecord(et2);
        
    //host waits until et2 has occured     
    hipEventSynchronize(et2);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, st2, et2);

    cout<<"Kernel time: "<<milliseconds<<"ms"<<endl;

    //copy data out
    hipMemcpy(host_out, dev_out, N * sizeof(double), hipMemcpyDeviceToHost);

    hipEventDestroy(st2);
    hipEventDestroy(et2);
}
    free(host_in);
    free(host_out);
    hipFree(dev_in);
    hipFree(dev_out);

  return 0;
}
